#include<stdio.h>
#include<iostream>
#include <unistd.h>
using namespace std;
void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}
void getGPUInfo()
{
    int count,i;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&count);
    for(i=0;i<count;i++)
    {
        hipGetDeviceProperties(&deviceProp,i);
        cout<<"GPU ID "<<i<<"\nGPU name is:"<<deviceProp.name<<"\nGPU total global memory is"
            <<deviceProp.totalGlobalMem/1024/1024<<" (MB)\n";
    }
    return;
}

int main()
{
        int i,dID;
        hipDeviceProp_t deviceProp;
        hipGetDeviceCount(&i);
        getGPUInfo();
        hipDeviceReset();
        return 0;
}
