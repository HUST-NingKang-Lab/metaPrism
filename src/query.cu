#include <iostream>
#include <fstream>
#include <queue>
#include <stdio.h>

#include "version.h"
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include ""
#include "meta_drive.cpp"
#include "meta_database.cpp"
#include "calc.cpp"
using namespace std;


int printhelp() {

    cout << "Welcome to Meta-Prism Beta " << endl;
    cout << "Version : " << Version << endl;
    cout << "Usage : " << endl;
    cout << "query_index [-option] value" << endl;
    cout << "\toption : " << endl;
    cout << "\t-l query sample list" << endl;
    cout << "\t-d database index path" << endl;
    cout << "\t-i query sample path" << endl;
    cout << "\t-o result output path, default is screen" << endl;
    cout << "\t-n hit number, default is 5" << endl;
    cout << "\t-e exhaustive search" << endl;
    cout << "\t-g to assign group number, default is off" << endl;
    cout << "\t-b biotype, default is none" << endl;
    cout << "\t-h help" << endl;
    exit(0);
    return 0;
};


int main(int argc, char *argv[]) {
    string indexname = "Default.txt";
    string queryfilename = "classification.txt";
    string outfilename = "";
    struct timeval tv_begin, tv_end;
    gettimeofday(&tv_begin, NULL);

    int t_number = 1;
    int is_index = 1;
    int r_number = 5;
    int group = -1;

    // temp
    //int gpulist[2]={0,1};


    string scroingtype = "F";
    string biotype = "none";
    string filterflag = "T";
    string filelist = "";
    //string tree_file;
    CompData* CD;
    CD=new CompData();
    Comp_init(CD);
    int i = 1;

    if (argc == 1)
        printhelp();

    while (i < argc) {
        if (argv[i][0] != '-') {
            printf("Argument # %d Error : Arguments must start with -\n", i);
            exit(0);
        };
        switch (argv[i][1]) {
            case 'd':
                indexname = argv[i + 1];
                break;
            case 'i':
                queryfilename = argv[i + 1];
                break;
            case 'e':
                is_index = 0;
                i--;
                break;
            case 'n':
                r_number = atoi(argv[i + 1]);
                break;
            case 'o':
                outfilename = argv[i + 1];
                break;
            case 'g':
                group = atoi(argv[i + 1]);
                break;
            case 'b':
                biotype = argv[i + 1];
                break;
            case 's':
                scroingtype = argv[i + 1];
                break;
            case 'f':
                filterflag = argv[i + 1];
                break;
            case 'l':
                filelist = argv[i + 1];
                break;
            default :
                printf("Unrec argument %s\n", argv[i]);
                printhelp();
                break;
        }
        i += 2;
    }
    hipGetDeviceCount(&deviceCount);
    if(deviceCount<1) {
        printf("no cuda device\n");
        exit(0);

    }
    calcID=new pthread_t[deviceCount];
    int ret;
    sem_init(&semse,0,0);
    sem_init(&semsf,0,100);
    sem_init(&semre,0,0);
    sem_init(&semrf,0,100);
    pthread_mutex_init(&sendLock,NULL);
    pthread_mutex_init(&calcLock,NULL);
    pthread_mutex_init(&resultLock,NULL);
    pthread_mutex_init(&trashLock,NULL);
    pthread_mutex_init(&killLock,NULL);
    pthread_mutex_init(&coutLock,NULL);
    //deviceCount=1;
    {
        cout<<"i'm loader pro\n";

        drivePara para;
        para.outpath=outfilename;
        para.filelist=filelist;
        para.indexname=indexname;
        para.n=r_number;
        para.t=t_number;
        para.group=group;
        para.scroingtype=scroingtype;
        para.filterflag=filelist;
        para.is_index=is_index;
        para.biotype=biotype;
        para.CD=CD;
        //drive(para);
        int m=0;
        int *deviceIds;
        deviceIds=new int[deviceCount];
        for(m=0;m<deviceCount;m++){
            deviceIds[m]=m;
            ret=pthread_create(&(calcID[m]),
                    NULL,calcSim,
                    (void*)(deviceIds+m));
            cout<<"creat one calc thread\n";
        }
        ret=pthread_create(&processID,NULL,process,(void*)&r_number);
        cout<<"creat one process thread\n";
        drive((void*)&para);

        gettimeofday(&tv_end, NULL);
        double time_use = double(tv_end.tv_sec - tv_begin.tv_sec) * 1000000 + double(tv_end.tv_usec - tv_begin.tv_usec);
        cout << "This query step costs: " << time_use / 1000000 << " s" << endl;
    }
    /*
    fpid=fork();
    if(fpid<0)
        printf("error in fork\n");
    else if (fpid==0){
        // loader
        cout<<"i'm loader pro\n";
        abdMap *pAbdMap;
        pAbdMap=new abdMap(indexname,CD);
        Meta_Database database(indexname, biotype, pAbdMap);
        drive(outfilename, filelist, indexname, r_number, t_number, group, scroingtype, filterflag, is_index,
                    database, is_gpu,CD);
    }
    else{
        bool device =false;
        int deviceID;
        for(i=0;i<deviceCount;i++){
            fpid=fork();
            if(fpid<0) printf("error in fork\n");
            else if(fpid==0){
                deviceID=i;
                device=true;
                break;
            }
        }
        if(device== false){
            cout<<"i'm waiter pro\n";
            //waiter
            exit(0);
        }
        else {
            cout<<"i'm executer pro "<<deviceID<<endl;
            //gpu executer
            exit(0);
        }
    }
     */
    return 0;
}
