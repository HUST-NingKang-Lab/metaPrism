#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <vector>
#include <sys/time.h>
#include "gpus.cu"


using namespace std;

#ifndef LeafN
#define LeafN 99322
#endif

#ifndef OrderN
#define OrderN 99321
#endif
#ifndef WarpSize
#define WarpSize 32
#endif
#ifndef ABD
#define ABD
struct Abd
{
    //char name[20];
    string name;
    float data[LeafN];
};
#endif

#ifndef CMPDATA
#define CMPDATA
struct CompData
{
    float Dist_1[OrderN];
    float Dist_2[OrderN];
    int Order_1[OrderN];
    int Order_2[OrderN];
    int Order_d[OrderN];
    int Id[LeafN];
};
#endif

#ifndef GPUC
#define GPUC
__global__ void gpu_Calc_sim(CompData *cd, Abd *v_Abd_1, Abd *v_Abd_2, float *results, int count);

class gpu_compare
{
public:
    gpu_compare();
    void init(int *GPUs,int count);
    int sendData(CompData * cd, Abd * abd1, Abd * abd2,int count, int version = 2);
    //please malloc or new memory space for these data, and don't free these memory untill you retrive the results
    //count means the number of Abd2
    //default to use second generation compare algorithm
    int act();
    int getResult(float * putResult);
    /*
    error table  0 for all right, 1 for no enough memory
    2 for error state for example, you ask the gpu_compare to act befor send data to it
    */
private:
    bool memFlag;
    int maxMem, deviceID, countCore,state,count;
    GPU * gpus;
    int gpuCount;
    int version,compFlag;
    int computSum;
};

gpu_compare::gpu_compare(){
    computSum=0;
    cout<<"using multi-GPU version\n";
}
void gpu_compare::init(int *GPUs, int gpuCount)
{
    GPU gpu;
    this->gpuCount=gpuCount;
    gpus=new GPU[gpuCount];
    for(int i=0;i<gpuCount;i++){
        gpus[i].init(GPUs[i]);
        computSum+=gpus[i].computSum;
    }
    this->state=1;
    memFlag= true;
}
int gpu_compare::sendData(CompData *cd, Abd *abd1, Abd *abd2,int count, int version )
{
    /*
    1 for no enough memory
    2 for state error
    */
    if (this->state!=1)
    {
        cout<<"gpu_compare is not in right state!\n";
        return 2;
    }
    this->count=count;
    int offset=0;
    for(int i=0;i<gpuCount;i++){
        gpus[i].count=count;
        gpus[i].alloc=gpus[i].computSum*(count/this->computSum+1);
        gpus[i].offset=offset;
        offset=gpus[i].alloc+offset;
    }
    for(int i=0;i<gpuCount;i++){
        gpus[i].sendData(cd,abd1,abd2);
        }
    
    this->state=2;
    return 0;
}

int gpu_compare::act()
{
    for(int i=0;i<gpuCount;i++){
        gpus[i].act();
    }
    this->state=3;
    return 0;
}

int gpu_compare::getResult(float *putResult)
{
    if(this->state!=3)
    {
        cout<<"gpu_compare state error\n";
        return 2;
    }
    for(int i=0;i<gpuCount;i++) {
        gpus[i].getResult(putResult);
        gpus[i].reset();
    }
    /*for(int i=0;i<count;i++)
    {
        cout<<i<<endl;
        printf("the similarity is: %f",putResult[i]);
    }*/
    this-> state=4;
    //putResult=this->results;

    this->state=1;
    return 0;
}
#endif
#ifndef calc
#define calc
__global__ void gpu_Calc_sim(CompData *cd, Abd *v_Abd_1, Abd *v_Abd_2, float *results,int count)
{
    //process memory data
    float *Dist_1;
    float *Dist_2;
    int *Order_1;
    int *Order_2;
    int *Order_d;
    int *Id;
    Dist_1 = cd->Dist_1;
    Dist_2 = cd->Dist_2;
    Order_1 = cd->Order_1;
    Order_2 = cd->Order_2;
    Order_d = cd->Order_d;
    Id = cd->Id;
    //change offset of each parameters

    int offset;
    offset=blockIdx.x*blockDim.x+threadIdx.x;
    if(offset>=count)
        return ;
    //const Meta_Result * buffer=(Meta_Result * )v_buffer+blockIdx.x;// don't know wether it's used
    v_Abd_2=(Abd *)v_Abd_2 +offset;
    const float *Abd_2 = v_Abd_2->data;
    const float *Abd_1 = v_Abd_1->data;

    //start origin data
    float Reg_1[70];
    float Reg_2[70];

    float total = 0;
    float total2=0;
    float total3=0;
    int root;

    for(int i = 0; i < OrderN; i++){
        //cout<<"i:"<<endl;

        int order_1 = Order_1[i];
        int order_2 = Order_2[i];
        int order_d = Order_d[i] + 70;

        float dist_1 = 1- Dist_1[i];
        float dist_2 = 1- Dist_2[i];

        float c1_1;
        float c1_2;

        float c2_1;
        float c2_2;


        if (order_1 >= 0){

            c1_1 = Abd_1[order_1];
            c1_2 = Abd_2[order_1];
        }
        else {
            c1_1 = Reg_1[order_1 + 70];
            c1_2 = Reg_2[order_1 + 70];
        }

        if (order_2 >= 0){

            c2_1 = Abd_1[order_2];
            c2_2 = Abd_2[order_2];

        }
        else {
            c2_1 = Reg_1[order_2 + 70];
            c2_2 = Reg_2[order_2 + 70];
        }
        //min
        float min_1 = (c1_1 < c1_2)?c1_1:c1_2;
        float min_2 = (c2_1 < c2_2)?c2_1:c2_2;

        total += min_1;
        total += min_2;

        //reduce
        Reg_1[order_d] = (c1_1 - min_1) * dist_1 + (c2_1 - min_2) * dist_2;
        Reg_2[order_d] = (c1_2 - min_1) * dist_1 + (c2_2 - min_2) * dist_2;
        root = order_d;
    }
    total += (Reg_1[root] < Reg_2[root])?Reg_1[root]:Reg_2[root];
    //cout<<"total:"<<total<<endl;
    //total2 += abs(Reg_1[root]-Reg_2[root]);
    //cout<<"second score:"<<total2<<endl;
    //cout<<"third score:"<<100-Reg_abs[root]<<endl;
    //cout<<total<<"\t"<<100-Reg_abs[root]<<endl;

    //return total;

    results[offset]=total;
    //100-Reg_abs[root];
    return;
}
#endif

