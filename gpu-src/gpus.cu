#include "hip/hip_runtime.h"

#include <iostream>
using namespace std;
#ifndef LeafN
#define LeafN 99322
#endif
#ifndef ABD
#define ABD
struct Abd
{
    //char name[20];
    string name;
    float data[LeafN];
};
#endif

#ifndef OrderN
#define OrderN 99321
#endif
#ifndef WarpSize
#define WarpSize 32
#endif
#ifndef CMPDATA
#define CMPDATA
struct CompData
{
    float Dist_1[OrderN];
    float Dist_2[OrderN];
    int Order_1[OrderN];
    int Order_2[OrderN];
    int Order_d[OrderN];
    int Id[LeafN];
};
#endif
__global__ void gpu_Calc_sim(CompData *cd, Abd *v_Abd_1, Abd *v_Abd_2, float *results,int count);


#ifndef gpu
#define gpu
class GPU{
public:
    int count,id,computSum;
    int alloc;
    int offset;
    hipDeviceProp_t deviceProp;
    void init(int i);
    void reset();
    int sendData(CompData *, Abd *, Abd *);
    int act();
    int getResult(float *putResult);
private:
    bool compFlag;
    struct Abd *abd1,*g_abd1,*abd2,*g_abd2;
    struct CompData *compData,*g_compData;
    float *results,*g_results;
};
void GPU::init(int i){
    this->id=i;
    hipGetDeviceProperties(&(this->deviceProp),id);
    computSum=deviceProp.warpSize*deviceProp.multiProcessorCount;
    alloc=offset=0;
    compFlag= true;
    return;
}
void GPU::reset(){
    alloc=offset=0;
    hipSetDevice(id);
    hipFree(g_abd1);
    hipFree(g_abd2);
    hipFree(g_results);
    abd1=abd2=g_abd1=g_abd2=0;g_results=0;
}
int GPU::sendData(CompData *cd, Abd *first_abd, Abd *second_abd){
    hipSetDevice(id);
    alloc=alloc+offset>count?count-offset:alloc;
    if (deviceProp.totalGlobalMem<sizeof(float)*(LeafN*(alloc+1)+alloc)*1.1)
        return 1;
    abd1=first_abd;
    abd2=second_abd+offset;
    hipMalloc((void **)&this->g_abd1,sizeof(Abd));
    hipMalloc((void **)&this->g_abd2,sizeof(Abd)*alloc);
    hipMalloc((void **)&this->g_results,sizeof(float)*alloc);
    if(this->compFlag) {
        hipMalloc((void **) &this->g_compData, sizeof(CompData));
        hipMemcpy(this->g_compData,cd,sizeof(CompData),hipMemcpyHostToDevice);
        compFlag= false;
    }
    hipMemcpy(this->g_abd1,abd1,sizeof(Abd),hipMemcpyHostToDevice);
    hipMemcpy(this->g_abd2,abd2,sizeof(Abd)*alloc,hipMemcpyHostToDevice);
    return 0;
}
int GPU::act(){
    hipSetDevice(id);
    int blocks;
    blocks=alloc/WarpSize+1;
    gpu_Calc_sim<<<blocks, WarpSize>>>(g_compData, g_abd1, g_abd2, g_results,alloc);
    return 0;
}
int GPU::getResult(float * putResult){
    putResult+=offset;
    hipSetDevice(id);
    hipMemcpy(putResult, this->g_results, sizeof(float) * this->alloc,hipMemcpyDeviceToHost);
    return 0;
}
#endif
#ifndef calc
#define calc
__global__ void gpu_Calc_sim(CompData *cd, Abd *v_Abd_1, Abd *v_Abd_2, float *results,int count)
{
    //process memory data
    float *Dist_1;
    float *Dist_2;
    int *Order_1;
    int *Order_2;
    int *Order_d;
    int *Id;
    Dist_1 = cd->Dist_1;
    Dist_2 = cd->Dist_2;
    Order_1 = cd->Order_1;
    Order_2 = cd->Order_2;
    Order_d = cd->Order_d;
    Id = cd->Id;
    //change offset of each parameters

    int offset;
    offset=blockIdx.x*blockDim.x+threadIdx.x;
    if(offset>=count)
        return ;
    //const Meta_Result * buffer=(Meta_Result * )v_buffer+blockIdx.x;// don't know wether it's used
    v_Abd_2=(Abd *)v_Abd_2 +offset;
    const float *Abd_2 = v_Abd_2->data;
    const float *Abd_1 = v_Abd_1->data;

    //start origin data
    float Reg_1[70];
    float Reg_2[70];

    float total = 0;
    float total2=0;
    float total3=0;
    int root;

    for(int i = 0; i < OrderN; i++){
        //cout<<"i:"<<endl;

        int order_1 = Order_1[i];
        int order_2 = Order_2[i];
        int order_d = Order_d[i] + 70;

        float dist_1 = 1- Dist_1[i];
        float dist_2 = 1- Dist_2[i];

        float c1_1;
        float c1_2;

        float c2_1;
        float c2_2;


        if (order_1 >= 0){

            c1_1 = Abd_1[order_1];
            c1_2 = Abd_2[order_1];
        }
        else {
            c1_1 = Reg_1[order_1 + 70];
            c1_2 = Reg_2[order_1 + 70];
        }

        if (order_2 >= 0){

            c2_1 = Abd_1[order_2];
            c2_2 = Abd_2[order_2];

        }
        else {
            c2_1 = Reg_1[order_2 + 70];
            c2_2 = Reg_2[order_2 + 70];
        }
        //min
        float min_1 = (c1_1 < c1_2)?c1_1:c1_2;
        float min_2 = (c2_1 < c2_2)?c2_1:c2_2;

        total += min_1;
        total += min_2;

        //reduce
        Reg_1[order_d] = (c1_1 - min_1) * dist_1 + (c2_1 - min_2) * dist_2;
        Reg_2[order_d] = (c1_2 - min_1) * dist_1 + (c2_2 - min_2) * dist_2;
        root = order_d;
    }
    total += (Reg_1[root] < Reg_2[root])?Reg_1[root]:Reg_2[root];
    //cout<<"total:"<<total<<endl;
    //total2 += abs(Reg_1[root]-Reg_2[root]);
    //cout<<"second score:"<<total2<<endl;
    //cout<<"third score:"<<100-Reg_abs[root]<<endl;
    //cout<<total<<"\t"<<100-Reg_abs[root]<<endl;

    //return total;

    results[offset]=total;
    //100-Reg_abs[root];
    return;
}
#endif

