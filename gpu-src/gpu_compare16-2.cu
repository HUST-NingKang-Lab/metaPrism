#include <iostream>

#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <sys/time.h>


using namespace std;

#ifndef LeafN
#define LeafN 99322
#endif

#ifndef OrderN
#define OrderN 99321
#endif

struct Abd
{
    //char name[20];
    string name;
    float data[LeafN];
};
struct CompData
{
    float Dist_1[OrderN];
    float Dist_2[OrderN];
    int Order_1[OrderN];
    int Order_2[OrderN];
    int Order_d[OrderN];
    int Id[LeafN];
};







#ifndef GPUC
#define GPUC

__global__ void gpu_Calc_sim(CompData *cd, Abd *v_Abd_1, Abd *v_Abd_2, float *results, int count);

class gpu_compare
{
public:
    gpu_compare();
    int sendData(CompData * cd, Abd * abd1, Abd * abd2,int count, int version = 2);
    //please malloc or new memory space for these data, and don't free these memory untill you retrive the results
    //count means the number of Abd2
    //default to use second generation compare algorithm
    int act();
    int getResult(float * putResult);
    /*
    error table  0 for all right, 1 for no enough memory
    2 for error state for example, you ask the gpu_compare to act befor send data to it
    */

private:
    int maxMem, deviceID, countCore,state,count;
    hipDeviceProp_t deviceProp;
    size_t freeMem, totalMem;
    CompData *compData, *g_compData;
    Abd *abd1, *abd2, *g_abd1, *g_abd2;
    float *results, *g_results;
    int version,compFlag;
};

gpu_compare::gpu_compare()
{
    hipChooseDevice(&this->deviceID, &this->deviceProp);
    hipGetDeviceProperties(&this->deviceProp, this->deviceID);
    this->maxMem = this->deviceProp.totalGlobalMem;
    this->countCore = this->deviceProp.multiProcessorCount;
    hipMemGetInfo(&freeMem, &totalMem);
    //cout<<"the best block size is : "<<this->bestBlockSize<<endl;
	//cout<<"the min Grid size is : "<<this->minGridSize<<endl;
    if(1)
    {
        cout<<"now using "<<this->deviceProp.name<<" device id:"<<this->deviceID<<endl;
        cout<<"count of SM is "<<this->deviceProp.multiProcessorCount<<endl;
        cout<<"warp size is (threads per SM) "<<this->deviceProp.warpSize<<endl;

    }
    this->compFlag=1;
    this->state=1;
}
int gpu_compare::sendData(CompData *cd, Abd *abd1, Abd *abd2,int count, int version )
{
    /*
    1 for no enough memory
    2 for state error
    */
    if (this->state!=1)
    {
        cout<<"gpu_compare is not in right state!\n";
        return 2;
    }
    this->count=count;
    this->compData = cd;
    this->abd1 = abd1;
    this->abd2 = abd2;
    this->version = version;
	
    if (totalMem<sizeof(float)*(LeafN*(count+1)+count)*1.1)
    {
        cout<<"no enough gpu memory, will update later to support it\n";
        return 1;
    }
    else
    {
        // malloc the memory
        hipMalloc((void **)&this->g_abd1,sizeof(Abd));
        hipMalloc((void **)&this->g_abd2,sizeof(Abd)*count);
        hipMalloc((void **)&this->g_results,sizeof(float)*count);
        if(this->compFlag) {
            hipMalloc((void **) &this->g_compData, sizeof(CompData));
            compFlag=0;
        }
    }
    //send memory from main mamory to GPU memory
    hipMemcpy(this->g_compData,cd,sizeof(CompData),hipMemcpyHostToDevice);
    hipMemcpy(this->g_abd1,abd1,sizeof(Abd),hipMemcpyHostToDevice);
    hipMemcpy(this->g_abd2,abd2,sizeof(Abd)*count,hipMemcpyHostToDevice);

    this->state=2;
    return 0;
}

int gpu_compare::act()
{
    int wrapSize,i,blocks,wrapCount;
    if(this->state!=2)
    {
        cout<<"gpu_compare state error\n";
        return 2;
    }
    blocks=this->count/16+1;
    gpu_Calc_sim<<<blocks, 16>>>(this->g_compData, this->g_abd1, this->g_abd2, this->g_results,this->count);
    this->state=3;
    return 0;
}

int gpu_compare::getResult(float *putResult)
{
    if(this->state!=3)
    {
        cout<<"gpu_compare state error\n";
        return 2;
    }
    this->results=putResult;
    //this->results=new float[count];
    hipMemcpy(this->results, this->g_results, sizeof(float) * this->count,hipMemcpyDeviceToHost);
    /*for(int i=0;i<count;i++)
    {
        cout<<i<<endl;
        printf("the similarity is: %f",putResult[i]);
    }*/
    this-> state=4;
    //putResult=this->results;

    hipFree(this->g_abd1);
    hipFree(this->g_abd2);
    hipFree(this->g_results);

    this->state=1;
    return 0;
}
__global__ void gpu_Calc_sim(CompData *cd, Abd *v_Abd_1, Abd *v_Abd_2, float *results,int count)
{
    //process memory data
    float *Dist_1;
    float *Dist_2;
    int *Order_1;
    int *Order_2;
    int *Order_d;
    int *Id;
    Dist_1 = cd->Dist_1;
    Dist_2 = cd->Dist_2;
    Order_1 = cd->Order_1;
    Order_2 = cd->Order_2;
    Order_d = cd->Order_d;
    Id = cd->Id;
    //change offset of each parameters

    int offset;
    offset=blockIdx.x*blockDim.x+threadIdx.x;
    if(offset>=count)
        return ;
    //const Meta_Result * buffer=(Meta_Result * )v_buffer+blockIdx.x;// don't know wether it's used
    v_Abd_2=(Abd *)v_Abd_2 +offset;
    const float *Abd_2 = v_Abd_2->data;
    const float *Abd_1 = v_Abd_1->data;

    //start origin data
    float Reg_1[70];
    float Reg_2[70];
    float Reg_abs[70];

    float total = 0;
    float total2=0;
    float total3=0;
    int root;

    for(int i = 0; i < OrderN; i++){
        //cout<<"i:"<<endl;

        int order_1 = Order_1[i];
        int order_2 = Order_2[i];
        int order_d = Order_d[i] + 70;

        float dist_1 = 1- Dist_1[i];
        float dist_2 = 1- Dist_2[i];

        float c1_1;
        float c1_2;

        float c2_1;
        float c2_2;

        float abs_1;
        float abs_2;

        if (order_1 >= 0){

            c1_1 = Abd_1[order_1];
            c1_2 = Abd_2[order_1];
            abs_1=abs(Abd_1[order_1]- Abd_2[order_1]) * 0.5;
        }
        else {
            c1_1 = Reg_1[order_1 + 70];
            c1_2 = Reg_2[order_1 + 70];
            abs_1=Reg_abs[order_1 + 70];
        }

        if (order_2 >= 0){

            c2_1 = Abd_1[order_2];
            c2_2 = Abd_2[order_2];
            abs_2=abs(Abd_1[order_2]-Abd_2[order_2]) * 0.5;

        }
        else {
            c2_1 = Reg_1[order_2 + 70];
            c2_2 = Reg_2[order_2 + 70];
            abs_2 = Reg_abs[order_2 + 70];
        }
        //min
        float min_1 = (c1_1 < c1_2)?c1_1:c1_2;
        float min_2 = (c2_1 < c2_2)?c2_1:c2_2;

        total += min_1;
        total2 += abs(c1_1-c1_2);


        total += min_2;
        total2 += abs(c2_1-c2_2);


        /*if(abs(c2_1-c2_2) !=0  || abs(c1_1-c1_2) !=0)
        {
        cout<<c1_1<<"-"<<c1_2<<"="<<c1_1-c1_2<<" "<<abs(c1_1-c1_2)<<endl;
        cout<<c2_1<<"-"<<c2_2<<"="<<c2_1-c2_2<<" "<<abs(c2_1-c2_2)<<endl;
        cout<<"total2:"<<total2<<endl;
        }
        */



        //reduce
        Reg_1[order_d] = (c1_1 - min_1) * dist_1 + (c2_1 - min_2) * dist_2;
        Reg_2[order_d] = (c1_2 - min_1) * dist_1 + (c2_2 - min_2) * dist_2;
        Reg_abs[order_d]= abs_1*dist_1 + abs_2*dist_2;

        root = order_d;
    }

    total += (Reg_1[root] < Reg_2[root])?Reg_1[root]:Reg_2[root];
    //cout<<"total:"<<total<<endl;
    //total2 += abs(Reg_1[root]-Reg_2[root]);
    //cout<<"second score:"<<total2<<endl;
    //cout<<"third score:"<<100-Reg_abs[root]<<endl;
    //cout<<total<<"\t"<<100-Reg_abs[root]<<endl;

    //return total;

    results[offset]=total;
    //100-Reg_abs[root];
    return;
}
#endif
