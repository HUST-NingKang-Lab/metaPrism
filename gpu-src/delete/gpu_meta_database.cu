#include "hip/hip_runtime.h"
//___________________copy from .cpp
#include <iostream>
#include <pthread.h>
#include "hip/hip_runtime.h"
#include ""
//_______________ copy from .h
#include <fstream>
#include <sstream>
#include <string>
#include <map>
#include <vector>

#include <sys/types.h>
#include <sys/dir.h>
#include <sys/stat.h>
#include <sys/wait.h>
#include <sys/mman.h>

#include <stdlib.h>
#include <unistd.h>

#include "index_entry.h"
//#include "tree_class.h"
#include "cfn.h"
#include "comp.h"

#ifndef META_DATABASE_H
#define META_DATABASE_H

#define Buffer_Size 1000
#define i_min 15
#define q_min 1
#define q_max 20
#define dom_t 60

using namespace std;

class Meta_Result
{

  public:
    friend class Meta_Database;

    Meta_Result()
    {
        m_value = 0;
        entry = 0;
    }
    Meta_Result(float _value, Index_Entry *_entry)
    {
        m_value = _value;
        entry = _entry;
    }

  private:
    float m_value;
    Index_Entry *entry;
};

class Meta_Database
{

  public:
    Meta_Database()
    {
        this->Entry_count = 0;
        this->Entry_number = 0;
        this->Key_dim = 0;
        this->Name = "";
        this->Abundance = 0;
        //Comp_init();
    }

    Meta_Database(string _name, int dim)
    {
        this->Entry_count = 0;
        this->Entry_number = 0;
        this->Key_dim = dim;
        this->Name = _name;
        this->Abundance = 0;
        //Comp_init();
    }

    Meta_Database(string _name, string path, int dim)
    {
        this->Entry_count = 0;
        this->Entry_number = 0;
        this->Key_dim = dim;
        this->Name = _name;
        this->Abundance = 0;
        this->Make_Index(path);
        //Comp_init();
    }
    Meta_Database(string infilename)
    {
        this->Entry_count = 0;
        this->Entry_number = 0;
        this->Abundance = 0;
        this->Load_Index(infilename);
        //Comp_init();
    }
    ~Meta_Database(){};

    void Insert_Index_by_Entry(Index_Entry *entry);
    void Insert_Group_by_Entry(Index_Entry *entry);

    void Insert_Index_by_File(string infilename, unsigned int group);

    int Make_Index(string path);
    int Make_Index_Add(string path, unsigned int group);

    //unsigned int Query(string infilename);
    //unsigned int Exhaustive_Query(string infilename);
    unsigned int Parallel_Indexed_Query(ostream &out, string infilename, int n, int t_number, int group);
    unsigned int Parallel_Exhaustive_Query(ostream &out, string infilename, int n, int t_number, int group);

    int Out_Index(string outfilename);
    int Load_Index(string infilename);
    int Merge_Index(string infilename);

    int Get_Entry_Count() { return this->Entry_count; };
    int Get_Key_Dim() { return this->Key_dim; };
    int Get_Key_Count() { return this->Meta_index.size(); };

    float Get_Abundance();
    float Update_Abundance();

  private:
    map<string, vector<Index_Entry *> *> Meta_index;
    map<unsigned int, vector<Index_Entry *> *> Meta_groups;
    int maxMem;
    int deviceID, countCore;
    hipDeviceProp_t deviceProp;
    unsigned int Entry_count;
    unsigned int Entry_number; //ID
    unsigned int Key_dim;
    string Name;

    //string Tree_file;

    int Abundance;
    //string Path;

    unsigned int Get_Entry_List(Index_Entry **list, int group);
    unsigned int Get_Entry_List(Index_Entry **list, string query_key, int group);

    unsigned int Parallel_Indexed_Query(string infilename, string query_key, Meta_Result *results, int n, int t_number, int group);

    unsigned int Parallel_Query(string infilename, Index_Entry **list, Meta_Result *results, int n, int t_number, int count);

    static void *Parallel_Query_Static(void *args);
};

struct Argument
{

    string infilename;
    Index_Entry **list;
    int thread;
    int per_number;
    int count;
    //string * tree_file;

    Meta_Result *buffer;
};

#endif
//_______________________________________________________________copy from .cpp
void Meta_Database::Insert_Index_by_Entry(Index_Entry *entry)
{

    if (this->Meta_index.count(entry->Get_key()) == 0)
        this->Meta_index[entry->Get_key()] = new vector<Index_Entry *>;

    this->Meta_index[entry->Get_key()]->push_back(entry);

    this->Entry_count++;

    if (entry->Get_abundance() > dom_t)
        this->Abundance++;
};

void Meta_Database::Insert_Group_by_Entry(Index_Entry *entry)
{

    if (this->Meta_groups.count(entry->Get_group()) == 0)
        this->Meta_groups[entry->Get_group()] = new vector<Index_Entry *>;

    this->Meta_groups[entry->Get_group()]->push_back(entry);
}

void Meta_Database::Insert_Index_by_File(string infilename, unsigned int group)
{

    string keys[this->Key_dim];
    float weight[this->Key_dim];

    Get_Key_Weight((infilename + "/classification.txt").c_str(), keys, weight, this->Key_dim);

    Index_Entry *entry = new Index_Entry(this->Entry_number, group, keys, weight, this->Key_dim, infilename, i_min);

    this->Entry_number++;

    this->Insert_Index_by_Entry(entry);

    this->Insert_Group_by_Entry(entry);
}

int Meta_Database::Make_Index(string path)
{

    //if (this->Meta_index.empty())
    //   this->Meta_index.clear();

    //this->Entry_count = 0;
    //this->Entry_number = 0;

    //this->Path = path;

    struct dirent *ent = NULL;
    DIR *pDir;
    pDir = opendir(path.c_str());
    if (pDir == NULL)
    {
        cerr << "Error: Cannot Open Path : " << path << endl;
        exit(0);
    }

    while ((ent = readdir(pDir)) != NULL)
        if ((ent->d_type == 4) && (ent->d_name[0] != '.'))
        {

            string inpathname = path + "/" + ent->d_name;

            string infilename = inpathname + "/classification.txt";
            ifstream infile(infilename.c_str(), ifstream::in);

            if (infile != NULL)
            {
                infile.close();
                infile.clear();
                //cout << inpathname << endl;
                this->Insert_Index_by_File(inpathname, 0);
            }
            else
                this->Make_Index(inpathname);
        }

    closedir(pDir);

    return this->Entry_count;
}

int Meta_Database::Make_Index_Add(string path, unsigned int group)
{

    struct dirent *ent = NULL;
    DIR *pDir;
    pDir = opendir(path.c_str());
    if (pDir == NULL)
    {
        cerr << "Error: Cannot Open Path : " << path << endl;
        exit(0);
    }

    unsigned int _entry_count = 0;

    while ((ent = readdir(pDir)) != NULL)
        if ((ent->d_type == 4) && (ent->d_name[0] != '.'))
        {

            string inpathname = path + "/" + ent->d_name;

            string infilename = inpathname + "/classification.txt";
            ifstream infile(infilename.c_str(), ifstream::in);

            if (infile != NULL)
            {
                infile.close();
                infile.clear();
                //cout << inpathname << endl;
                this->Insert_Index_by_File(inpathname, group);
                _entry_count++;
            }
            else
                _entry_count += this->Make_Index_Add(inpathname, group);
        }
    closedir(pDir);
    return _entry_count;
}

unsigned int Meta_Database::Get_Entry_List(Index_Entry **list, int group)
{

    unsigned int count = 0;

    map<string, vector<Index_Entry *> *>::iterator map_iter;

    for (map_iter = this->Meta_index.begin(); map_iter != this->Meta_index.end(); map_iter++)
    {

        vector<Index_Entry *>::iterator vect_iter;
        for (vect_iter = map_iter->second->begin(); vect_iter != map_iter->second->end(); vect_iter++)
            if (((*vect_iter)->Get_group() == group) || (group < 0))
            {

                list[count] = *vect_iter;

                count++;
            }
    }

    return count;
}

unsigned int Meta_Database::Get_Entry_List(Index_Entry **list, string query_key, int group)
{

    unsigned int count = 0;

    if (this->Meta_index.count(query_key) == 0)
        return 0;

    vector<Index_Entry *>::iterator vect_iter;

    for (vect_iter = this->Meta_index[query_key]->begin(); vect_iter != this->Meta_index[query_key]->end(); vect_iter++)
        if (((*vect_iter)->Get_group() == group) || (group < 0))
        {

            list[count] = *vect_iter;

            count++;
        }

    return count;
}
/*
void *Meta_Database::Parallel_Query_Static(void *args, void *datas)
{
    //datas added by kk
    string infilename = ((Argument *)args)->infilename;
    Index_Entry **list = ((Argument *)args)->list;
    int thread = ((Argument *)args)->thread;
    int per_number = ((Argument *)args)->per_number;
    int count = ((Argument *)args)->count;
    //string * tree_file = ((Argument *)args)->tree_file;
    //change to gpu at here
    Meta_Result *buffer = ((Argument *)args)->buffer;

    for (int i = 0; i < per_number; i++)
    { //Loop in each thread
        int sam_file = thread * per_number + i;
        if (sam_file >= count)
            break;

        string entry_filename = list[sam_file]->Get_filename() + "/classification.txt";
        
        //cout << (float)(j + 1) / (float)(per_number) * 100.0 << "%\tfinished" << endl;
    }

    pthread_exit(0);
}
*/

unsigned int Meta_Database::Parallel_Query(string infilename, Index_Entry **list, Meta_Result *results, int n, int t_number, int count)
{
    //change to gpu at here
    //count means the sum of compare to be done
    //calculate the proper t_number for GPU
    string entry_filename;
    size_t mem, totalMem;
    int i;
    Meta_Result *buffer, *g_buffer;
    //Argument *args,*g_args;
    //pthread_t *t,*g_t;
    CompData *compData, *g_compData;
    float *Abd_1, *Abd_2, *g_Abd_1, *g_Abd_2;
    float * g_results;
    hipChooseDevice(&this->deviceID, &this->deviceProp);
    hipGetDeviceProperties(&this->deviceProp, this->deviceID);
    this->maxMem = this->deviceProp.totalGlobalMem;
    this->countCore = this->deviceProp.multiProcessorCount;
    //guess mem is 1MB per thread
    hipMemGetInfo(&mem, &totalMem);
    if (mem / 1024 / 1024 > count)
    {
        t_number = count;
        hipMalloc((void **)&g_buffer, sizeof(Meta_Result) * count);
        //hipMalloc((void **)&g_args,sizeof(Argument)*1);
        //hipMalloc((void **)&g_t,sizeof(pthread_t)*count);
        hipMalloc((void **)&g_compData, sizeof(CompData) * 1);
        hipMalloc((void **)&g_Abd_1, sizeof(float) * 1 * LeafN);
        hipMalloc((void **)&g_Abd_2, sizeof(float) * count * LeafN);
        hipMalloc((void **)&g_results, sizeof(float) * count);
    }
    else
    { //if don't have enough memory
        cout << "GPU Memory out of range, havn't got solution for this\n";
    }
    // malloc CPU Memory to send to GPU memory
    buffer = new Meta_Result[1];
    //args=(Argument*)malloc(sizeof(Argument)*1);//just malloc one arg
    //t=*(pthread_t*)malloc(t_number*sizeof(pthread_t));
    compData = new CompData[1];
    Abd_1 = new float[LeafN];
    Abd_2 = new float[LeafN * count];
    float *gpu_results = new float[count];

    get_compData(compData);
    /*
        args[0].infilename = infilename;
        args[0].list = list;
        args[0].thread = 0;
        args[0].per_number = count%cores==0 ? count/cores : count/cores+1;
        args[0].count = count;
        args[0].buffer = buffer;
        */
    // don't use Static any more, because the limitation of memory swaping is memory speed, not CPU speed
    Load_abd(infilename.c_str(), compData->Id, Abd_1);
    for (i = 0; i < count; i++)
    {
        entry_filename = list[i]->Get_filename() + "/classification.txt";
        Load_abd(entry_filename.c_str(), compData->Id, Abd_2 + i * LeafN);
    }
    

    hipMemcpy(g_compData, compData, sizeof(CompData), hipMemcpyHostToDevice);
    hipMemcpy(g_buffer, buffer, sizeof(Meta_Result) * count, hipMemcpyHostToDevice);
    hipMemcpy(g_Abd_1, Abd_1, sizeof(float) * 1 * LeafN, hipMemcpyHostToDevice);
    hipMemcpy(g_Abd_2, Abd_2, sizeof(float) * count * LeafN, hipMemcpyHostToDevice);

    //prepare for GPU calculation
    //start calculation

    //need optimize
    gpu_Calc_sim<<<count, 1>>>(g_compData, g_Abd_1, g_Abd_2, g_results);

        //paste back
        hipMemcpy(gpu_results, g_results, sizeof(float) * count,hipMemcpyDeviceToHost);
        /*
        hipMemcpy(compData,g_compData,sizeof(CompData),hipMemcpyDeviceToHost);
        hipMemcpy(buffer,g_buffer,sizeof(Meta_Result)*count,hipMemcpyDeviceToHost);
        hipMemcpy(Abd_1,g_Abd_1,sizeof(float)*1*LeafN,hipMemcpyDeviceToHost);
        hipMemcpy(Abd_2,g_Abd_2,sizeof(float)*count*LeafN,hipMemcpyDeviceToHost);
        */

        //process the data
        for (i = 0; i < count; i++)
    {
        buffer[i].m_value = gpu_results[i];
        buffer[i].entry = list[i];
    }
    hipFree(g_compData);
    hipFree(g_buffer);
    hipFree(g_Abd_1);
    hipFree(g_Abd_2);

    //__stop, following is old version
    for (int i = 0; i < count; i++)
        if (buffer[i].m_value > results[n - 1].m_value)
        {
            results[n - 1] = buffer[i];
            for (int j = n - 1; j > 0; j--)
                if (results[j].m_value > results[j - 1].m_value)
                {
                    Meta_Result temp = results[j];
                    results[j] = results[j - 1];
                    results[j - 1] = temp;
                }
                else
                    break;
        }

    if (n < count)
        return n;
    else
        return count;
}

unsigned int Meta_Database::Parallel_Indexed_Query(string infilename, string query_key, Meta_Result *results, int n, int t_number, int group)
{

    unsigned int count = 0;

    if (this->Meta_index.count(query_key) == 0)
        return 0;

    else
        count = this->Meta_index[query_key]->size();

    //To get the list of entry
    Index_Entry *list[count];

    count = this->Get_Entry_List(list, query_key, group);

    //for (int i = 0; i < count; i++)
    //  cout << list[i]->Get_filename() << endl;

    n = this->Parallel_Query(infilename, list, results, n, t_number, count);

    return n;
}

unsigned int Meta_Database::Parallel_Indexed_Query(ostream &out, string infilename, int n, int t_number, int group)
{

    string keys[this->Key_dim];

    float weight[this->Key_dim];

    Get_Key_Weight(infilename.c_str(), keys, weight, this->Key_dim);

    Meta_Result results[n];

    int n_count = 0;

    for (int i = this->Key_dim; i > 0; i--)
        if (weight[i - 1] >= q_min)
        {

            string temp_keys[i];
            float temp_weight[i];

            for (int j = 0; j < i; j++)
            {

                temp_keys[j] = keys[j];
                temp_weight[j] = weight[j];
            }

            for (int j = 0; j < i - 1; j++)
                for (int k = j + 1; k < i; k++)
                    if ((temp_keys[j] > temp_keys[k]) && (abs(temp_weight[j] - temp_weight[k]) < delta))
                    {

                        string temp_k = temp_keys[j];
                        temp_keys[j] = temp_keys[k];
                        temp_keys[k] = temp_k;

                        float temp_w = temp_weight[j];
                        temp_weight[j] = temp_weight[k];
                        temp_weight[k] = temp_w;
                    }

            string query_key = "";

            for (int j = 0; j < i; j++)
                query_key += temp_keys[j];

            n_count += this->Parallel_Indexed_Query(infilename, query_key, results, n, t_number, group);

            //cout << i << "\t" << n_count << endl;

            if (weight[i - 1] >= q_max)
                break;
        }

    if (n_count == 0)
    {

        out << "#Meta-Storms" << endl;
        out << "#Not hits" << endl;

        return 0;
    }

    n = (n < n_count) ? n : n_count;

    for (int i = 0; i < n; i++)
    {

        out << "#Meta-Storm" << endl;
        out << "Match " << i + 1 << endl;
        out << "Similarity: " << results[i].m_value << "%" << endl;
        out << "Sample Group: " << results[i].entry->Get_group() << endl;
        out << "Sample path: " << results[i].entry->Get_filename() << endl;
    }

    return n;
}

unsigned int Meta_Database::Parallel_Exhaustive_Query(ostream &out, string infilename, int n, int t_number, int group)
{

    unsigned int count = 0;

    if (group < 0)
        count = this->Entry_count;
    else if (this->Meta_groups.count(group) > 0)
        count = this->Meta_groups[group]->size();

    if (count == 0)
    {

        out << "#Meta-Storms" << endl;
        out << "#Not hits" << endl;

        return 0;
    }
    //To get the list of entry
    Index_Entry *list[count];

    this->Get_Entry_List(list, group);

    Meta_Result results[n];

    n = this->Parallel_Query(infilename, list, results, n, t_number, count);

    for (int i = 0; i < n; i++)
    {

        out << "#Meta-Storm" << endl;
        out << "Match " << i + 1 << endl;
        out << "Similarity: " << results[i].m_value << "%" << endl;
        out << "Sample Group: " << results[i].entry->Get_group() << endl;
        out << "Sample path: " << results[i].entry->Get_filename() << endl;
    }

    return n;
}

int Meta_Database::Out_Index(string outfilename)
{

    mkdir(outfilename.c_str(), 0755);
    string indexfile = outfilename + "/index.txt";
    string nametable = outfilename + "/nametable.txt";

    ofstream out_index(indexfile.c_str(), ofstream::out);
    if (!out_index)
    {
        cerr << "Error : Cannot Open Index File : " << indexfile << endl;
        exit(0);
    }

    ofstream out_name(nametable.c_str(), ofstream::out);
    if (!out_name)
    {
        cerr << "Error : Cannot Open Name Table : " << nametable << endl;
        exit(0);
    }

    map<string, vector<Index_Entry *> *>::iterator miter = this->Meta_index.begin();

    out_index << "Database : " << this->Name << "\t"
              << "Total Entry Count : " << this->Entry_count << "\t"
              << "Total Key Count : " << this->Meta_index.size() << "\t"
              << "Key Dim : " << this->Key_dim << "\t"
              << "Total Group Count : " << this->Meta_groups.size() << endl;

    out_name << "Database : " << this->Name << "\t"
             << "Total Entry Count : " << this->Entry_count << "\t"
             << "Total Key Count : " << this->Meta_index.size() << "\t"
             << "Key Dim : " << this->Key_dim << "\t"
             << "Total Group Count : " << this->Meta_groups.size() << endl;
    out_name << "Databasae ID\tGroup ID\tPath\tTop phyla abundence" << endl;

    while (miter != this->Meta_index.end())
    {
        out_index << ">" << miter->first << endl;
        vector<Index_Entry *>::iterator viter;
        for (viter = miter->second->begin(); viter != miter->second->end(); viter++)
        {
            out_index << (*viter)->Get_id() << "\t";
            out_name << (*viter)->Get_id() << "\t" << (*viter)->Get_group() << "\t" << (*viter)->Get_filename();
            for (int i = 0; i < this->Key_dim; i++)
                out_name << "\t" << (*viter)->Get_key(i) << "\t" << (*viter)->Get_weight(i);
            out_name << endl;
        }
        out_index << endl;

        miter++;
    }

    out_index.close();
    out_index.clear();
    out_name.close();
    out_name.clear();

    return this->Entry_count;
}

int Meta_Database::Load_Index(string infilename)
{

    string indexfile = infilename + "/index.txt";
    string nametable = infilename + "/nametable.txt";

    ifstream in_index(indexfile.c_str(), ifstream::in);
    if (!in_index)
    {
        cerr << "Error: Cannot Open Database Index File : " << indexfile << endl;
        exit(0);
    }
    ifstream in_name(nametable.c_str(), ifstream::in);
    if (!in_name)
    {
        cerr << "Error: Cannot Open Database Name Table : " << nametable << endl;
        exit(0);
    }

    if (this->Meta_index.empty())
        this->Meta_index.clear();

    map<unsigned int, Index_Entry *> entry_map;

    string buffer;

    //Nametable

    //Label
    getline(in_name, buffer);

    //DB Name;
    int begin = buffer.find("Database : ") + 11;
    int end = buffer.find("\t");
    this->Name = buffer.substr(begin, end - begin);
    //Key dim;
    begin = buffer.find("Key Dim : ") + 10;
    end = buffer.size();
    this->Key_dim = atoi((buffer.substr(begin, end - begin)).c_str());
    //Entry_count;
    this->Entry_count = 0;
    //Abundance
    this->Abundance = 0;

    //Label
    getline(in_name, buffer);

    while (getline(in_name, buffer))
    {

        if (buffer.size() == 0)
            continue;

        unsigned int id;
        unsigned int group;
        string filename;

        string keys[this->Key_dim];
        float weight[this->Key_dim];

        int begin = 0;
        int end = buffer.find('\t', begin);

        id = atoi(buffer.substr(begin, end - begin).c_str());

        begin = end + 1;
        end = buffer.find('\t', begin);

        group = atoi(buffer.substr(begin, end - begin).c_str());

        begin = end + 1;
        end = buffer.find('\t', begin);

        filename = buffer.substr(begin, end - begin);

        for (int i = 0; i < this->Key_dim; i++)
        {

            begin = end + 1;
            end = buffer.find('\t', begin);
            keys[i] = buffer.substr(begin, end - begin);

            begin = end + 1;
            end = buffer.find('\t', begin);
            weight[i] = atof(buffer.substr(begin, end - begin).c_str());
        }

        Index_Entry *entry = new Index_Entry(id, group, keys, weight, this->Key_dim, filename, i_min);

        entry_map[id] = entry;

        if (id >= this->Entry_number)
            this->Entry_number = id + 1;
        /*
          istringstream strin(buffer);
          unsigned int id;
          string filename;
          strin >> id >> filename;
          
          namemap[id] = filename;
          */
    }
    //Index

    getline(in_index, buffer);

    while (getline(in_index, buffer))
    {

        if (buffer.size() == 0)
            continue;
        if (buffer[0] == '>')
        {
            string key = buffer.substr(1, buffer.size() - 1);
            getline(in_index, buffer);
            istringstream strin(buffer);
            unsigned int id;
            while (strin >> id)
            {
                if (entry_map.count(id) == 0)
                {

                    cerr << "Error : Cannot find Sample ID : " << id << endl;
                    exit(0);
                }

                this->Insert_Index_by_Entry(entry_map[id]);
                this->Insert_Group_by_Entry(entry_map[id]);
            }
        }
    }
    in_index.close();
    in_index.clear();
    in_name.close();
    in_name.clear();
    return this->Entry_count;
}

float Meta_Database::Get_Abundance()
{

    return (float)this->Abundance / (float)this->Entry_count;
}

float Meta_Database::Update_Abundance()
{

    unsigned int temp_abundance = 0;

    unsigned int count = this->Entry_count;

    Index_Entry *list[count];

    this->Get_Entry_List(list, -1);

    for (int i = 0; i < count; i++)
        if (list[i]->Get_abundance() >= dom_t)
            temp_abundance++;
    //else cout << list[i]->Get_abundance() << endl;

    this->Abundance = temp_abundance;

    return (float)temp_abundance / (float)count;
}
